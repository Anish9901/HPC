#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void addVectors(int* A, int* B, int* C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    int n = 10;
    int *A, *B, *C;
    int size = n * sizeof(int);
    int *devA, *devB, *devC;
    hipMalloc(&devA, size);
    hipMalloc(&devB, size);
    hipMalloc(&devC, size);

    hipHostMalloc(&A, size, hipHostMallocDefault);
    hipHostMalloc(&B, size, hipHostMallocDefault);
    hipHostMalloc(&C, size, hipHostMallocDefault);

    for (int i = 0; i < n; i++) 
    {
        A[i] = i;
        cout<< A[i];
        B[i] = i * 2;
        cout<< B[i];
    }
    hipMemcpy(devA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, size, hipMemcpyHostToDevice);

    addVectors<<< 256,n >>>(devA, devB, devC, n);
    hipMemcpy(devC, C, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < 10; i++)
    {
        cout<< C[i] << " ";
    }
    cout<< "\n";
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    return 0;
}
/* __global__ void addVectors(int* A, int* B, int* C, int n) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i < n) {
C[i] = A[i] + B[i];
}
}
int main() {
int n = 1000000;
int* A, * B, * C;
int size = n * sizeof(int);
// Allocate memory on the host
cudaMallocHost(&A, size);
cudaMallocHost(&B, size);
cudaMallocHost(&C, size);
// Initialize the vectors
for (int i = 0; i < n; i++) {
A[i] = i;
B[i] = i * 2;
}
50
// Allocate memory on the device
int* dev_A, * dev_B, * dev_C;
cudaMalloc(&dev_A, size);
cudaMalloc(&dev_B, size);
cudaMalloc(&dev_C, size);
// Copy data from host to device
cudaMemcpy(dev_A, A, size,
cudaMemcpyHostToDevice);
cudaMemcpy(dev_B, B, size,
cudaMemcpyHostToDevice);
// Launch the kernel
int blockSize = 256;
int numBlocks = (n + blockSize
- 1) / blockSize;
51
// Copy data from device to host
cudaMemcpy(C, dev_C, size, cudaMemcpyDeviceToHost);
// Print the results
for (int i = 0; i < 10; i++) {
cout << C[i] << " ";
}
cout << endl;
// Free memory
cudaFree(dev_A);
cudaFree(dev_B);
cudaFree(dev_C);
cudaFreeHost(A);
cudaFreeHost(B);
cudaFreeHost(C);
return 0; */